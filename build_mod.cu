#include "NvInfer.h"
#include "NvOnnxParser.h"
#include "logger.h"
#include "common.h"
#include<memory>
// #include "buffers.h"
// #include "utils/preprocess.h"
#include "gflags/gflags.h"
#include <NvInferPlugin.h>

#include <opencv2/opencv.hpp>

using namespace nvinfer1;

// define input flags

DEFINE_string(onnx_file, "smoke_dla34.onnx", "onnx file path");
DEFINE_string(calib_dir, "", "calibration data dir");
DEFINE_string(calib_list_file, "", "calibration data list file");
DEFINE_string(input_name, "input", "network input name");
DEFINE_int32(input_h, 384, "network input height");
DEFINE_int32(input_w, 1280, "network input width");
DEFINE_int32(input_c, 3, "network input channel");
DEFINE_bool(int8, false, "use int8 mode");
DEFINE_string(model_name, "smoke", "model name");
DEFINE_string(format, "nchw", "input format");


int main(int argc, char **argv)
{
    if (argc < 2)
    {
        // std::cout << "Usage: ./build [onnx_file] [calib_dir] [calib_list_file]" << std::endl;
        return -1;
    }

    gflags::ParseCommandLineFlags(&argc, &argv, true);

    const char *onnx_file_path = FLAGS_onnx_file.c_str();
    const char *calib_dir = FLAGS_calib_dir.c_str();
    const char *calib_list_file = FLAGS_calib_list_file.c_str();

    int input_h = FLAGS_input_h;
    int input_w = FLAGS_input_w;
    const char *input_name = FLAGS_input_name.c_str();

    bool useInt8 = FLAGS_int8;

    // remove extension of onnx_file_path
    std::string output_file_name = FLAGS_onnx_file.substr(0, FLAGS_onnx_file.find_last_of(".")) + ".engine";

    sample::gLogger.setReportableSeverity(nvinfer1::ILogger::Severity::kVERBOSE);

    initLibNvInferPlugins(&sample::gLogger.getTRTLogger(), "");

    // 1. Create builder
    // auto builder = SampleUniquePtr<nvinfer1::IBuilder>(nvinfer1::createInferBuilder(sample::gLogger.getTRTLogger()));
    nvinfer1::IBuilder* builder = nvinfer1::createInferBuilder(sample::gLogger.getTRTLogger());
    if (!builder)
    {
        return -1;
    }


    // 2. Create network
    const auto explicitBatch = 1U << static_cast<uint32_t>(nvinfer1::NetworkDefinitionCreationFlag::kEXPLICIT_BATCH);
    nvinfer1::INetworkDefinition* network = builder->createNetworkV2(explicitBatch);
    if (!network)
    {
        return -1;
    }

    // 3. Create builder config
    nvinfer1::IBuilderConfig *config=builder->createBuilderConfig();
    if (!config)
    {
        return -1;
    }

    // 4. Create ONNX Parser
    nvonnxparser::IParser* parser = nvonnxparser::createParser(*network, sample::gLogger.getTRTLogger());

    // 5. Parse ONNX model
    auto parsed = parser->parseFromFile(onnx_file_path, static_cast<int>(sample::gLogger.getReportableSeverity()));
    if (!parsed)
    {
        return -1;
    }

    auto input = network->getInput(0);
    auto profile = builder->createOptimizationProfile();
    if (FLAGS_format == "nchw")
    {
        profile->setDimensions(input->getName(), nvinfer1::OptProfileSelector::kMIN, nvinfer1::Dims4{1, FLAGS_input_c, input_h, input_w});
        profile->setDimensions(input->getName(), nvinfer1::OptProfileSelector::kOPT, nvinfer1::Dims4{1, FLAGS_input_c, input_h, input_w});
        profile->setDimensions(input->getName(), nvinfer1::OptProfileSelector::kMAX, nvinfer1::Dims4{1, FLAGS_input_c, input_h, input_w});
    }
    else
    {
        profile->setDimensions(input->getName(), nvinfer1::OptProfileSelector::kMIN, nvinfer1::Dims4{1, input_h, input_w, FLAGS_input_c});
        profile->setDimensions(input->getName(), nvinfer1::OptProfileSelector::kOPT, nvinfer1::Dims4{1, input_h, input_w, FLAGS_input_c});
        profile->setDimensions(input->getName(), nvinfer1::OptProfileSelector::kMAX, nvinfer1::Dims4{1, input_h, input_w, FLAGS_input_c});
    }
    config->addOptimizationProfile(profile);

    // 6. set calibration configuration
    if (builder->platformHasFastFp16())
    {
        config->setFlag(nvinfer1::BuilderFlag::kFP16);
    }
    builder->setMaxBatchSize(1);
    config->setMaxWorkspaceSize(1 << 30);
    

    // 7. Create CUDA stream for profiling
    hipStream_t profileStream;
    hipError_t cudaStatus = hipStreamCreate(&profileStream);
    if (cudaStatus != hipSuccess)
    {
        return -1;
    }

    config->setProfileStream(profileStream);

    // 8. Build Serialized Engine
    nvinfer1::ICudaEngine *plan=builder->buildEngineWithConfig(*network, *config);
    if (!plan)
    {
        return -1;
    }

        //序列化engine
    nvinfer1::IHostMemory *engineString =plan->serialize();
    if (engineString == nullptr || engineString->size() == 0)
    {
        std::cout << "Failed building serialized engine!" << std::endl;
        return 1;
    }
    std::cout << "Succeeded building serialized engine!" << std::endl;

    // 9. save engine
    std::ofstream engine_file(output_file_name, std::ios::binary);
    engine_file.write((char *)engineString->data(), engineString->size());
    engine_file.close();

    builder->destroy();
    network->destroy();


    return 0;
}


