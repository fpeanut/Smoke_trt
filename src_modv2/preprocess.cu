#include "hip/hip_runtime.h"
#include "preprocess.h"

#include <hip/hip_runtime_api.h>

namespace apollo {
namespace perception {
namespace camera {

#ifndef CUDA_CHECK
#define CUDA_CHECK(callstr)                                                                    \
    {                                                                                          \
        hipError_t error_code = callstr;                                                      \
        if (error_code != hipSuccess)                                                         \
        {                                                                                      \
            std::cerr << "CUDA error " << error_code << " at " << __FILE__ << ":" << __LINE__; \
            assert(0);                                                                         \
        }                                                                                      \
    }
#endif // CUDA_CHECK

static uint8_t *img_buffer_device = nullptr;

struct AffineMatrix
{
    float value[6];
};

void cuda_preprocess_init(int max_image_size)
{
    // prepare input data in device memory
    CUDA_CHECK(hipMalloc((void **)&img_buffer_device, max_image_size * 3));
}

void cuda_preprocess_destroy()
{
    CUDA_CHECK(hipFree(img_buffer_device));
}

// 一个线程处理一个像素点
__global__ void preprocess_kernel(
    uint8_t *src, float *dst, int dst_width,
    int dst_height, int edge)
{
    int position = blockDim.x * blockIdx.x + threadIdx.x;
    if (position >= edge)
        return;

    int dx = position % dst_width; // 计算当前线程对应的目标图像的x坐标
    int dy = position / dst_width; // 计算当前线程对应的目标图像的y坐标

    // normalization（对原图中(x,y)坐标的像素点3个通道进行归一化）
    float c0 = src[dy * dst_width * 3 + dx * 3 + 0];
    float c1 = src[dy * dst_width * 3 + dx * 3 + 1];
    float c2 = src[dy * dst_width * 3 + dx * 3 + 2];

    // bgr to rgb
    float t = c2;
    c2 = c0;
    c0 = t;

    // rgbrgbrgb to rrrgggbbb
    // NHWC to NCHW
    int area = dst_width * dst_height;
    float *pdst_c0 = dst + dy * dst_width + dx;
    float *pdst_c1 = pdst_c0 + area;
    float *pdst_c2 = pdst_c1 + area;
    // *pdst_c0 = c0;
    // *pdst_c1 = c1;
    // *pdst_c2 = c2;
    *pdst_c0 = (c0-123.675)/58.395;
    *pdst_c1 = (c1-116.28)/57.12;
    *pdst_c2 = (c2-103.53)/57.375;
}

// GPU做归一化、BGR2RGB、NHWC to NCHW
void cuda_preprocess(
    uint8_t *src, float *dst, int dst_width, int dst_height)
{

    int img_size = dst_width * dst_height * 3;
    CUDA_CHECK(hipMemcpy(img_buffer_device, src, img_size, hipMemcpyHostToDevice));

    // 一个线程处理一个像素点，一共需要 dst_height * dst_width 个线程
    int jobs = dst_height * dst_width;
    int threads = 256;
    int blocks = ceil(jobs / (float)threads);


    preprocess_kernel<<<blocks, threads>>>(
        img_buffer_device, dst, dst_width, dst_height, jobs);
}

void SMOKEPreprocess(cv::Mat &img, int inputW, int inputH, float *buffer)
{
    cuda_preprocess(img.ptr(), buffer, inputW, inputH);
}

}  // namespace camera
}  // namespace perception
}  // namespace apollo
